#include "hip/hip_runtime.h"
#include <algorithm>
#include "Constants.h"
#include <stdio.h>
#include "ProjHelperFunCuda.h"
#include "ProjHelperFun.h"
#include "CudaUtilProj.cu.h"
#include "CudaKernels.cu.h"

// divide and round up
unsigned int divup(unsigned int a, unsigned int b) {
    return ((a + (b - 1)) / b);
}

void
rollback( const unsigned            outer, 
          const unsigned            g,   
          const PrivGlobsInvCuda&   globsinvCuda,
          ExpGlobsCuda&             globsCuda,
          TempGlobsCuda&            temp_globs,
          unsigned                  numX,
          unsigned                  numY ) {

    //unsigned int struct_size = outer*numX*numY*sizeof(REAL);

    /********************************
            CUDA Declarations
    *********************************/
    REAL *d_u = temp_globs.d_u;
    REAL *d_u_t = temp_globs.d_u_t;
    REAL *d_v = temp_globs.d_v;
    REAL *d_a = temp_globs.d_a;
    REAL *d_b = temp_globs.d_b;
    REAL *d_c = temp_globs.d_c;
    REAL *d_y = temp_globs.d_y;
    REAL *d_yy = temp_globs.d_yy;
    //REAL *d_abc = temp_globs.d_abc;

    /********************************
            CUDA Kernel calls
    *********************************/
    dim3 threads_per_3d_block(32,32,1); //32*32 = 1024 threads total
    dim3 num_3d_blocks(divup(numX, threads_per_3d_block.x), divup(numY, threads_per_3d_block.y), divup(outer, threads_per_3d_block.z));
    dim3 num_3d_blocks_interchange(divup(numY, threads_per_3d_block.y), divup(numX, threads_per_3d_block.x), divup(outer, threads_per_3d_block.z));
    
    ExplicitXNaiveCuda<<<num_3d_blocks,threads_per_3d_block>>>(globsCuda.d_myVarX, globsCuda.d_myResult, globsinvCuda.d_myDxx, d_u_t, globsinvCuda.d_myTimeline, g, numX, numY, outer);
    hipDeviceSynchronize();

    // TODO: Transpose u
    transpose<REAL, 32>( d_u_t, d_u, numX, numY, outer);

    hipError_t cudaReturnCode = hipPeekAtLastError();
    if(cudaReturnCode != hipSuccess ) 
    {
        printf("\nCUDA ERROR: \"%i: %s\".\n", cudaReturnCode, hipGetErrorString(cudaReturnCode));
    }

    // TODO: loop interchange
    ExplicitYNaiveCuda<<<num_3d_blocks_interchange,threads_per_3d_block>>>(globsCuda.d_myVarY, globsCuda.d_myResult, globsinvCuda.d_myDyy, d_u, d_v, numX, numY, outer);
    hipDeviceSynchronize();

    // TODO: Transpose myVarX
    //ImplicitXSetupCubeCuda<<<num_3d_blocks,threads_per_3d_block>>>(globsCuda.d_myVarX, globsinvCuda.d_myDxx, d_abc, globsinvCuda.d_myTimeline, g, numX, numY, outer);
    ImplicitXSetupNaiveCuda<<<num_3d_blocks,threads_per_3d_block>>>(globsCuda.d_myVarX, globsinvCuda.d_myDxx, d_a, d_b, d_c, globsinvCuda.d_myTimeline, g, numX, numY, outer);
    hipDeviceSynchronize();

    dim3 tridag_y_block_size(1024, 1, 1);
    dim3 tridag_y_blocks(divup(numY, tridag_y_block_size.x), divup(outer, tridag_y_block_size.y), 1); 

    //tridagCubeCuda<<<tridag_y_blocks, tridag_y_block_size>>>(d_abc, d_u, numX, numY, outer, d_u, d_yy);
    //tridagNaiveCuda<<<tridag_y_blocks, tridag_y_block_size>>>(d_a, d_b, d_c, d_u, numX, numY, outer, d_u, d_yy);
    dim3 tridag_fb_block_size2(32, 32, 1);
    dim3 tridag_fb_blocks2(divup(numY, tridag_fb_block_size2.x), divup(outer, tridag_fb_block_size2.y), 1); 
    for(int k=1; k<numX; k++) { 
        tridagCoalCudaForward2<<<tridag_fb_blocks2, tridag_fb_block_size2>>>(d_a, d_b, d_c, d_u, numY, numX, outer, k, d_u, d_yy);
        hipDeviceSynchronize();
    }
    
    for(int k=numX-1; k>=0; k--) {
        tridagCoalCudaBackward2<<<tridag_fb_blocks2, tridag_fb_block_size2>>>(d_a, d_b, d_c, d_u, numY, numX, outer, k, d_u, d_yy);
        hipDeviceSynchronize();
    }
    //tridagCoalCuda<<<tridag_y_blocks, tridag_y_block_size>>>(d_a, d_b, d_c, d_u, numX, numY, outer, d_u, d_yy);
    hipDeviceSynchronize();
    exit(0);
    // TODO: transpose u (we have already done it up there for ExplicitXNaiveCuda)
    transpose<REAL, 32>( d_u, d_u_t, numY, numX, outer);
    ImplicitYSetupNaiveCuda<<<num_3d_blocks,threads_per_3d_block>>>(globsCuda.d_myVarY, globsinvCuda.d_myDyy, d_a, d_b, d_c, d_u_t, d_v, d_y, globsinvCuda.d_myTimeline, g, numX, numY, outer);
    //ImplicitYSetupCubeCuda<<<num_3d_blocks,threads_per_3d_block>>>(globsCuda.d_myVarY, globsinvCuda.d_myDyy, d_abc, d_u_t, d_v, d_y, globsinvCuda.d_myTimeline, g, numX, numY, outer);
    hipDeviceSynchronize();

    dim3 tridag_fb_block_size(32, 32, 1);
    dim3 tridag_fb_blocks(divup(numX, tridag_fb_block_size.x), divup(outer, tridag_fb_block_size.y), 1); 

    dim3 tridag_x_block_size(1024, 1);
    dim3 tridag_x_blocks(divup(numX, tridag_x_block_size.x), divup(outer, tridag_x_block_size.y), 1); 

    //tridagCubeCuda<<<tridag_x_blocks, tridag_x_block_size>>>(d_abc, d_y, numY, numX, outer, globsCuda.d_myResult, d_yy);
    tridagNaiveCuda<<<tridag_x_blocks, tridag_x_block_size>>>(d_a, d_b, d_c, d_y, numY, numX, outer, globsCuda.d_myResult, d_yy);
    // for(int k=1; k<numY; k++) { 
    //     tridagCoalCudaForward<<<tridag_fb_blocks, tridag_fb_block_size>>>(d_a, d_b, d_c, d_y, numY, numX, outer, k, globsCuda.d_myResult, d_yy);
    //     hipDeviceSynchronize();
    // }
    
    // for(int k=numY-1; k>=0; k--) {
    //     tridagCoalCudaBackward<<<tridag_fb_blocks, tridag_fb_block_size>>>(d_a, d_b, d_c, d_y, numY, numX, outer, k, globsCuda.d_myResult, d_yy);
    //     hipDeviceSynchronize();
    // }
}

void   run_OrigCPU(  
                const unsigned int&   outer,
                const unsigned int&   numX,
                const unsigned int&   numY,
                const unsigned int&   numT,
                const REAL&           s0,
                const REAL&           t, 
                const REAL&           alpha, 
                const REAL&           nu, 
                const REAL&           beta,
                      REAL*           res   // [outer] RESULT
) {
    printf("Dimensions: numX: %i, numY: %i, numT: %i, outer: %i\n", numX, numY, numT, outer);

    PrivGlobsInvCuda globs_inv_cuda(numX, numY, numT);
    unsigned int threads_per_block = 32; //32*32 = 1024 threads total
    unsigned int num_blocksX = divup(numX, threads_per_block);
    unsigned int num_blocksY = divup(numY, threads_per_block);
    unsigned int num_blocksT = divup(numT, threads_per_block);

    InitGlobsInvX<<<num_blocksX, threads_per_block>>>(globs_inv_cuda.d_myX, globs_inv_cuda.d_myXindex, numX, s0, alpha, nu, t);
    InitGlobsInvY<<<num_blocksY, threads_per_block>>>(globs_inv_cuda.d_myY, globs_inv_cuda.d_myYindex, numY, s0, alpha, nu, t);
    InitGlobsInvT<<<num_blocksT, threads_per_block>>>(globs_inv_cuda.d_myTimeline, numT, t);
    hipDeviceSynchronize();
    InitOperatorCuda<<<num_blocksX, threads_per_block>>>(globs_inv_cuda.d_myX, globs_inv_cuda.d_myDxx, numX);
    InitOperatorCuda<<<num_blocksY, threads_per_block>>>(globs_inv_cuda.d_myY, globs_inv_cuda.d_myDyy, numY);
    hipDeviceSynchronize();

    ExpGlobsCuda globs_cuda(numX, numY, numT);

    TempGlobsCuda temp_globs(numX, numY, numT);

    dim3 threads_per_3d_block(32,32,1); //32*32 = 1024 threads total
    dim3 num_3d_blocks(divup(numX, threads_per_3d_block.x), divup(numY, threads_per_3d_block.y), divup(outer, threads_per_3d_block.z));
    dim3 num_3d_blocks_test(divup(numY, threads_per_3d_block.y), divup(numX, threads_per_3d_block.x), divup(outer, threads_per_3d_block.z));
    SetPayoffCuda<<<num_3d_blocks_test,threads_per_3d_block>>>(globs_inv_cuda.d_myX, globs_cuda.d_myResult, 0.001, numX, numY, outer);
    hipDeviceSynchronize();   

    // why this interchange is safe to do
    // why this loop cannot be parallelized
    for(int g = numT-2; g>=0; --g) {

        UpdateParamsCuda<<<num_3d_blocks_test,threads_per_3d_block>>>(
            globs_inv_cuda.d_myX, globs_inv_cuda.d_myY, globs_inv_cuda.d_myTimeline,
            globs_cuda.d_myVarX, globs_cuda.d_myVarY, 
            alpha, beta, nu, g, numX, numY, outer);
        hipDeviceSynchronize();

        // rollback only reads the value of i and globs but does not modify
        rollback(outer, g, globs_inv_cuda, globs_cuda, temp_globs, numX, numY);
    }
    
    REAL* d_globResults;
    hipMalloc((void**)&d_globResults, sizeof(REAL) * outer);
    GetResults<<<divup(outer, threads_per_block), threads_per_block>>>(d_globResults, globs_cuda.d_myResult
        , globs_inv_cuda.d_myXindex, globs_inv_cuda.d_myYindex
        , numX, numY, outer);
    hipDeviceSynchronize();

    hipMemcpy(res, d_globResults, sizeof(REAL) * outer, hipMemcpyDeviceToHost);

    hipFree(d_globResults);
}

