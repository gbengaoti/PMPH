#include "hip/hip_runtime.h"
#include <algorithm>
#include "Constants.h"
#include <stdio.h>
#include "ProjHelperFunCuda.h"
#include "ProjHelperFun.h"
#include "CudaUtilProj.cu.h"
#include "CudaKernels.cu.h"

// divide and round up
unsigned int divup(unsigned int a, unsigned int b) {
    return ((a + (b - 1)) / b);
}

// return a value that is integer divisible by aligner and not larger
// than the input
unsigned int alignNum(unsigned int numToAlign, unsigned int aligner) {
    return aligner  * (numToAlign - numToAlign % aligner) / aligner;
}


void
rollback( const unsigned            outer, 
          const unsigned            g,   
          const PrivGlobsInvCuda&   globsinvCuda,
          ExpGlobsCuda&             globsCuda,
          TempGlobsCuda&            temp_globs,
          unsigned                  numX,
          unsigned                  numY ) {

    /********************************
            CUDA Declarations
    *********************************/
    REAL *d_u = temp_globs.d_u;
    REAL *d_u_t = temp_globs.d_u_t;
    REAL *d_v = temp_globs.d_v;
    REAL *d_a = temp_globs.d_a;
    REAL *d_b = temp_globs.d_b;
    REAL *d_c = temp_globs.d_c;
    REAL *d_y = temp_globs.d_y;
    REAL *d_yy = temp_globs.d_yy;

    /********************************
            CUDA Kernel calls
    *********************************/
    dim3 threads_per_3d_block(32,32,1); //32*32 = 1024 threads total
    dim3 num_3d_blocks(divup(numX, threads_per_3d_block.x), divup(numY, threads_per_3d_block.y), divup(outer, threads_per_3d_block.z));
    dim3 num_3d_blocks_interchange(divup(numY, threads_per_3d_block.y), divup(numX, threads_per_3d_block.x), divup(outer, threads_per_3d_block.z));
    
    ExplicitXNaiveCuda<<<num_3d_blocks,threads_per_3d_block>>>(globsCuda.d_myVarX, globsCuda.d_myResult, globsinvCuda.d_myDxx, d_u_t, globsinvCuda.d_myTimeline, g, numX, numY, outer);
    hipDeviceSynchronize();

    transpose<REAL, 32>( d_u_t, d_u, numX, numY, outer);
    ExplicitYNaiveCuda<<<num_3d_blocks_interchange,threads_per_3d_block>>>(globsCuda.d_myVarY, globsCuda.d_myResult, globsinvCuda.d_myDyy, d_u, d_v, numX, numY, outer);
    hipDeviceSynchronize();

    ImplicitXSetupNaiveCuda<<<num_3d_blocks,threads_per_3d_block>>>(globsCuda.d_myVarX, globsinvCuda.d_myDxx, d_a, d_b, d_c, globsinvCuda.d_myTimeline, g, numX, numY, outer);
    hipDeviceSynchronize();

    // set up shared memory tridag - this is an attempt to optimize the 
    // number of threads and shared memory usage
    unsigned int maxSharedMemory = 48 * 1024;
    unsigned int maxNumRowsX = maxSharedMemory / (sizeof(REAL) * numX);
    maxNumRowsX = min<unsigned int>(128u, maxNumRowsX);
    maxNumRowsX = alignNum(maxNumRowsX, 16u);
    if (maxNumRowsX == 0)
    {
        maxNumRowsX = 1;
    }

    dim3 tridag_y_block_size(maxNumRowsX, 1, 1);
    dim3 tridag_y_blocks(divup(numY, tridag_y_block_size.x), divup(outer, tridag_y_block_size.y), 1); 

    tridagShMemCuda<<<tridag_y_blocks, tridag_y_block_size, (maxNumRowsX+1) * numX * sizeof(REAL)>>>(d_a, d_b, d_c, d_u, numX, numY, outer, d_u, d_yy);
    hipDeviceSynchronize();

    transpose<REAL, 32>( d_u, d_u_t, numY, numX, outer);
    ImplicitYSetupNaiveCuda<<<num_3d_blocks,threads_per_3d_block>>>(globsCuda.d_myVarY, globsinvCuda.d_myDyy, d_a, d_b, d_c, d_u_t, d_v, d_y, globsinvCuda.d_myTimeline, g, numX, numY, outer);
    hipDeviceSynchronize();

    // set up shared memory tridag - this is an attempt to optimize the 
    // number of threads and shared memory usage
    unsigned int maxNumRowsY = maxSharedMemory / (sizeof(REAL) * numY);
    maxNumRowsY = min<unsigned int>(128u, maxNumRowsY);
    maxNumRowsY = alignNum(maxNumRowsY, 16u);
    if (maxNumRowsY == 0)
    {
        maxNumRowsY = 1;
    }
    dim3 tridag_x_block_size(maxNumRowsY, 1, 1);
    dim3 tridag_x_blocks(divup(numX, tridag_x_block_size.x), divup(outer, tridag_x_block_size.y), 1); 
    tridagShMemCuda<<<tridag_x_blocks, tridag_x_block_size, (maxNumRowsY+1) * numY * sizeof(REAL)>>>(d_a, d_b, d_c, d_y, numY, numX, outer, globsCuda.d_myResult, d_yy);
    hipDeviceSynchronize();
}

void   run_OrigCPU(  
                const unsigned int&   outer,
                const unsigned int&   numX,
                const unsigned int&   numY,
                const unsigned int&   numT,
                const REAL&           s0,
                const REAL&           t, 
                const REAL&           alpha, 
                const REAL&           nu, 
                const REAL&           beta,
                      REAL*           res   // [outer] RESULT
) {
    printf("Dimensions: numX: %i, numY: %i, numT: %i, outer: %i\n", numX, numY, numT, outer);

    PrivGlobsInvCuda globs_inv_cuda(numX, numY, numT);
    unsigned int threads_per_block = 32; //32*32 = 1024 threads total
    unsigned int num_blocksX = divup(numX, threads_per_block);
    unsigned int num_blocksY = divup(numY, threads_per_block);
    unsigned int num_blocksT = divup(numT, threads_per_block);

    InitGlobsInvX<<<num_blocksX, threads_per_block>>>(globs_inv_cuda.d_myX, globs_inv_cuda.d_myXindex, numX, s0, alpha, nu, t);
    InitGlobsInvY<<<num_blocksY, threads_per_block>>>(globs_inv_cuda.d_myY, globs_inv_cuda.d_myYindex, numY, s0, alpha, nu, t);
    InitGlobsInvT<<<num_blocksT, threads_per_block>>>(globs_inv_cuda.d_myTimeline, numT, t);
    hipDeviceSynchronize();
    InitOperatorCuda<<<num_blocksX, threads_per_block>>>(globs_inv_cuda.d_myX, globs_inv_cuda.d_myDxx, numX);
    InitOperatorCuda<<<num_blocksY, threads_per_block>>>(globs_inv_cuda.d_myY, globs_inv_cuda.d_myDyy, numY);
    hipDeviceSynchronize();

    ExpGlobsCuda globs_cuda(numX, numY, numT);

    TempGlobsCuda temp_globs(numX, numY, numT);

    dim3 threads_per_3d_block(32,32,1); //32*32 = 1024 threads total
    dim3 num_3d_blocks(divup(numX, threads_per_3d_block.x), divup(numY, threads_per_3d_block.y), divup(outer, threads_per_3d_block.z));
    SetPayoffCuda<<<num_3d_blocks,threads_per_3d_block>>>(globs_inv_cuda.d_myX, globs_cuda.d_myResult, 0.001, numX, numY, outer);
    hipDeviceSynchronize();   

    // why this interchange is safe to do
    // why this loop cannot be parallelized
    for(int g = numT-2; g>=0; --g) {

        UpdateParamsCuda<<<num_3d_blocks,threads_per_3d_block>>>(
            globs_inv_cuda.d_myX, globs_inv_cuda.d_myY, globs_inv_cuda.d_myTimeline,
            globs_cuda.d_myVarX, globs_cuda.d_myVarY, 
            alpha, beta, nu, g, numX, numY, outer);
        hipDeviceSynchronize();

        // rollback only reads the value of i and globs but does not modify
        rollback(outer, g, globs_inv_cuda, globs_cuda, temp_globs, numX, numY);
    }
    
    REAL* d_globResults;
    hipMalloc((void**)&d_globResults, sizeof(REAL) * outer);
    GetResults<<<divup(outer, threads_per_block), threads_per_block>>>(d_globResults, globs_cuda.d_myResult
        , globs_inv_cuda.d_myXindex, globs_inv_cuda.d_myYindex
        , numX, numY, outer);
    hipDeviceSynchronize();

    hipMemcpy(res, d_globResults, sizeof(REAL) * outer, hipMemcpyDeviceToHost);

    hipFree(d_globResults);

    // did we encounter any errors?
    hipError_t cudaReturnCode = hipPeekAtLastError();
    if(cudaReturnCode != hipSuccess ) 
    {
        printf("\nCUDA ERROR: \"%i: %s\".\n", cudaReturnCode, hipGetErrorString(cudaReturnCode));
    }
}

